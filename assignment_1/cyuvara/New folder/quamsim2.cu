
#include <stdio.h>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
__global__ void
quamsim(const float *U, const float *A, float *B, const int *Q, int number_elem)
{    int i = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ int base_memory_idx ;
    __shared__ float A_shared[64];
    int memory_idx;
    int Q_bit_vector=0;
    int Q_bit_mask ;
    if(threadIdx.x==0){
        base_memory_idx = blockIdx.x;
        for(int it=0;it<6;it++)
           base_memory_idx = ((base_memory_idx >> Q[it])<<(Q[it]+1)) | ((1<< Q[it])-1)& base_memory_idx;     }
    __syncthreads();
    Q_bit_mask = threadIdx.x;
    for(int it=0;it<5;it++){
      if((Q_bit_mask & 1)== 1)
       Q_bit_vector =  Q_bit_vector | 1<<Q[it];
      else
       Q_bit_vector =  Q_bit_vector & ~(1<<Q[it]);  
      Q_bit_mask = Q_bit_mask >>1;    }
    memory_idx = base_memory_idx | Q_bit_vector; 
    A_shared[threadIdx.x] = A[memory_idx]; 
    A_shared[threadIdx.x|1<<5] = A[memory_idx | 1<<Q[5]];
    __syncthreads();
    if (i < number_elem)
    {        for(int it=0;it<6;it++){         
           int index = ((threadIdx.x >> it)<<(it+1)) | ((1<<it)-1)& threadIdx.x;   
           float temp =A_shared[index] ;
           A_shared[index]= (U[it*4]*A_shared[index]) + (U[it*4+1]*A_shared[index ^ (1<<it)]);
           A_shared[index ^ (1<<it)] =  (U[it*4+2]*temp) + (U[it*4+3]*A_shared[index ^ (1<<it)]);
         __syncthreads();
        } 
        B[memory_idx]=A_shared[threadIdx.x];
        B[memory_idx| 1<<Q[5]] = A_shared[threadIdx.x|1<<5];          }}

void CallKernelFunction(float *d_U,float *d_A,float *d_B,int *d_Q,int threadsPerBlock, int blocksPerGrid,int number_elem);
int main(int argc, char** argv)
{    hipError_t err = hipSuccess;
    vector<float> A_number;
    float num;
    int count=0;
    float h_U[24];
    size_t size ;
    int number_elem,q_bit[6]; 
    FILE *myfile;
    myfile = fopen(argv[1],"r"); 
    while(fscanf(myfile, "%f", &num) == 1) { 
      A_number.push_back(num);
      count++;    }    
    number_elem = A_number.size()-30;
    size = number_elem*sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    if (h_A == NULL || h_B == NULL)
    {        exit(EXIT_FAILURE);    }
    for(int i=0;i<A_number.size()-6;i++)
      if(i<24) h_U[i]=A_number[i];
      else h_A[i-24]=A_number[i];

    q_bit[0] = A_number[A_number.size()-6];
    q_bit[1] = A_number[A_number.size()-5];
    q_bit[2] = A_number[A_number.size()-4];
    q_bit[3] = A_number[A_number.size()-3];
    q_bit[4] = A_number[A_number.size()-2];
    q_bit[5] = A_number[A_number.size()-1];
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    float *d_U = NULL;
    err = hipMalloc((void **)&d_U, 24*sizeof(float));
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    int *d_Q = NULL;
    err = hipMalloc((void **)&d_Q, 6*sizeof(int));
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    err = hipMemcpy(d_Q,q_bit, 6*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    err = hipMemcpy(d_U, h_U, 24*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    int threadsPerBlock = 32;
    int blocksPerGrid =(number_elem>>1 + threadsPerBlock - 1) / threadsPerBlock;
    CallKernelFunction(d_U,d_A,d_B,d_Q,threadsPerBlock,blocksPerGrid,number_elem);    
    err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    for (int i = 0; i < number_elem; ++i)
     printf("%0.3f\n",h_B[i]);
    err = hipFree(d_A);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    err = hipFree(d_B);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    err = hipFree(d_U);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }
    err = hipFree(d_Q);
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    }

    free(h_A);
    free(h_B);
    err = hipDeviceReset();
    if (err != hipSuccess)
    {       exit(EXIT_FAILURE);    }
    return 0;}

void CallKernelFunction(float *d_U,float *d_A,float *d_B,int *d_Q,int threadsPerBlock, int blocksPerGrid,int number_elem){
    hipError_t err;    
    quamsim<<<blocksPerGrid, threadsPerBlock>>>(d_U,d_A,d_B,d_Q, number_elem>>1);
    err = hipGetLastError();
    if (err != hipSuccess)
    {        exit(EXIT_FAILURE);    } }

